#ifndef KERNEL
#define KERNEL

//////////////////////////////////////////////////////////
A lattice Boltzmann fluid flow solver written using CUDA
Code adapted from Bruce Jones LBM-C by Erhardt Markus
////////////////////////////////////////////////////////////////////////////////
//
// D2Q9 Lattice configuration:
//
//       6   2   5
//        \  |  /
//         \ | /
//          \|/
//       3---0---1
//          /|\
//         / | \
//        /  |  \
//       7   4   8
//
///////////////////////////////////////////////////////////////////////////////


#include <stdio.h>
#include "./data_types.cuh"
#include "./macros.cu"
#include "./solver.cu"
#include "./prepareCuda.cuh"
#include "model_builder.cu"
#include "cuda_util.cu"

// Include THRUST libraries
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/device_vector.h>

// DEVICE VARIABLE DECLARATION
Lattice *lattice_device;
Domain *domain_device;
DomainConstant *domain_constants_device;
OutputController *output_controller_device;

// HOST VARIABLE DECLARATION
Lattice *lattice_host, *lattice_device_prototype;
Domain *domain_host;
DomainConstant *domain_constants_host;
OutputController *output_controller_host;
Timing *times;
ProjectStrings *project;
ModelBuilder model_builder;



// EXECUTES ALL ROUTINES REQUIRED FOR THE MODEL SET UP

void setup(char *data_file)
{
	// Set cuda device to use
	hipSetDevice(0);
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(iterate_kernel), hipFuncCachePreferL1);
	
	// Allocate container structures

	combi_malloc<Lattice>(&latsetuptice_host, &lattice_device, sizeof(Lattice)); //combi_malloc (T **host_pointer, T **device_pointer, size_t size) into cuda_util.cu
	combi_malloc<Domain>(&domain_host, &domain_device, sizeof(Domain));
	combi_malloc<DomainConstant>(&domain_constants_host, &domain_constants_device, sizeof(DomainConstant));
	combi_malloc<OutputController>(&output_controller_host, &output_controller_device, sizeof(OutputController));
	domain_constants_host = (DomainConstant *)malloc(sizeof(DomainConstant));
	times = (Timing *)malloc(sizeof(Timing));
	project = (ProjectStrings *)malloc(sizeof(ProjectStrings));
	lattice_device_prototype = (Lattice *)malloc(sizeof(Lattice));

	ModelBuilder tmpmb(data_file, lattice_host, lattice_device,
		domain_constants_host, domain_constants_device,
		domain_host, domain_device,
		output_controller_host, output_controller_device,
		times, project);
	model_builder = tmpmb;

	int z_len = 1;
	#if DIM > 2
		z_len = domain_constants_host->length[2];
	#endif
}





